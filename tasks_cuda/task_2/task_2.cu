#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <iostream>
#include <fstream>
#define Num_elements 20
#define Num_threads_x 5
#define Num_threads_y 5
using namespace std;
__global__ void addKernel(int *a, int *b, int *c)
{
	int i = blockIdx.x*blockDim.x + threadIdx.x;
	int j = blockIdx.y*blockDim.y + threadIdx.y;
	c[i*Num_elements+j] = a[i*Num_elements+j] + b[i*Num_elements+j];
}

void add(int *a, int *b, int *c)
{
	int *dev_a;
	int *dev_b;
	int *dev_c;
	hipMalloc((void**)&dev_a, Num_elements * Num_elements * sizeof(int));
	hipMalloc((void**)&dev_b, Num_elements * Num_elements * sizeof(int));
	hipMalloc((void**)&dev_c, Num_elements * Num_elements * sizeof(int));
	hipMemcpy(dev_a, a, Num_elements * Num_elements * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(dev_b, b, Num_elements * Num_elements * sizeof(int), hipMemcpyHostToDevice);
	dim3 blocks(Num_elements / Num_threads_x, Num_elements / Num_threads_y);
	dim3 threads(Num_threads_x, Num_threads_y);
	addKernel << <blocks, threads>> > (dev_a, dev_b, dev_c);
	hipDeviceSynchronize();
	hipMemcpy(c, dev_c, Num_elements * Num_elements * sizeof(int), hipMemcpyDeviceToHost);
}



int main()
{
	ofstream out;
	out.open("C:\\tasks_cuda\\task_2.txt", ios::out);
	int *a = new int[Num_elements*Num_elements];
	int *b = new int[Num_elements*Num_elements];
	int *c = new int[Num_elements*Num_elements];

	for (int i = 0; i < Num_elements; i++) {
		for (int j = 0; j < Num_elements; j++) {
			a[i*Num_elements+j] = -i;
			b[i*Num_elements+j] = j;
		}

	}

	add(a, b, c);
	if (out.is_open()) {
		for (int i = 0; i < Num_elements; i++) {
			for (int j = 0; j < Num_elements; j++)
				out << c[i*Num_elements + j] << ' ';
			out << '\n';
		}
	}
	delete[] a;
	delete[] b;
	delete[] c;
	return 0;
}


