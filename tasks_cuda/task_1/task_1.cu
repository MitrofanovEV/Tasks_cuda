#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <fstream>
#include <iostream>
#define N 10
using namespace std;
__global__ void addKernel(int *a, int *b, int *c)
{
	int i = blockIdx.x;
	//int i = threadIdx.x;
	c[i] = a[i] + b[i];
}

void add(int *a, int *b, int *c)
{
	int *dev_a = 0;
	int *dev_b = 0;
	int *dev_c = 0;
	hipMalloc((void**)&dev_a, N * sizeof(int));
	hipMalloc((void**)&dev_b, N * sizeof(int));
	hipMalloc((void**)&dev_c, N * sizeof(int));
	hipMemcpy(dev_a, a, N * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(dev_b, b, N * sizeof(int), hipMemcpyHostToDevice);
	addKernel << <N, 1 >> > (dev_a, dev_b, dev_c);
	//addKernel << <1, N >> > (dev_a, dev_b, dev_c);
	hipDeviceSynchronize();
	hipMemcpy(c, dev_c, N * sizeof(int), hipMemcpyDeviceToHost);
}



int main()
{
	ofstream out;
	out.open("C:\\tasks_cuda\\task_1.txt", ios::out);
	int a[N], b[N], c[N];
	for (int i = 0; i < N; i++) {
		a[i] = -i;
		b[i] = i * i;
	}

	add(a, b, c);
	if (out.is_open()) {
		for (int i = 0; i < N; i++) {
			out << a[i] << ' ' << b[i] << ' ' << c[i] <<'\n';
		}
	}
	else
		cout << "error";
	return 0;
}


