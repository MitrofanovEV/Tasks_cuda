#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <iostream>
#include <fstream>
#define Num_elements 10000
#define Num_threads 100
using namespace std;
__global__ void addKernel(int *a, int *b, int *c)
{
	int i = blockIdx.x*blockDim.x + threadIdx.x;
	c[i] = a[i] + b[i];
}

void add(int *a, int *b, int *c)
{
	int *dev_a = 0;
	int *dev_b = 0;
	int *dev_c = 0;
	hipMalloc((void**)&dev_a, Num_elements * sizeof(int));
	hipMalloc((void**)&dev_b, Num_elements * sizeof(int));
	hipMalloc((void**)&dev_c, Num_elements * sizeof(int));
	hipMemcpy(dev_a, a, Num_elements * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(dev_b, b, Num_elements * sizeof(int), hipMemcpyHostToDevice);
	addKernel << <(Num_elements + (Num_threads - 1))/ Num_threads, Num_threads >> > (dev_a, dev_b, dev_c);
	hipDeviceSynchronize();
	hipMemcpy(c, dev_c, Num_elements * sizeof(int), hipMemcpyDeviceToHost);
}



int main()
{
	ofstream out;
	out.open("C:\\tasks_cuda\\task_1_2.txt", ios::out);
	int a[Num_elements], b[Num_elements], c[Num_elements];

	for (int i = 0; i < Num_elements; i++) {
		a[i] = -i;
		b[i] = i * i;
	}

	add(a, b, c);
	if (out.is_open()) {
		for (int i = 0; i < Num_elements; i++) {
			out << a[i] << ' ' << b[i] << ' ' << c[i] << '\n';
		}
	}

	return 0;
}


